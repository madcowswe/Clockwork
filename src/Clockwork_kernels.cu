// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "hip/device_functions.h"

const int N = 7;
const int blocksize = 7;

__global__
	void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}


int testcuda()
{
	char a[N] = "Hello ";
	int b[N] = {15, 10, 6, 0, -11, 1, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc( (void**)&ad, csize );
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );

	printf("%s\n", a);
	return 0;
}

template <unsigned N>
__global__ void Clockwork(uint32_t* staticbank,
						  uint32_t* regbank,
						  uint32_t* sharedbank1,
						  uint32_t* sharedbank2,
						  //uint32_t N,
						  int* bestiBuff,
						  int* bestiBuffHead)
{
	unsigned threadID = blockIdx.x * blockDim.x + threadIdx.x;

	uint32_t xstatic = staticbank[threadID];

	__shared__ uint32_t sb1[N];
	__shared__ uint32_t sb2[N];

	for (int i = threadIdx.x; i < N; i += blockDim.x)
	{
		sb1[i] = sharedbank1[i];
		sb2[i] = sharedbank2[i];
	}

	for (int rbidx = 0; rbidx < N; rbidx += 8)
	{
		uint32_t rb[8];
		for (int i = 0; i < 8; i++)
		{
			rb[i] = regbank[rbidx + i];
		}

		for (int i = 0; i < N; i++)
		{
			uint32_t acc1 = xstatic ^ sb2[i];
			for (int j = 0; j < N; j++)
			{
				uint32_t acc2 = acc1 ^ sb1[i];
				for (int k = 0; k < 8; k++)
				{
					//Only bother cheking for perfect xor, same as equal
					if (acc2 == rb[i])
					{
						int storeloc = atomicAdd(bestiBuffHead, 4);
						bestiBuff[storeloc] = j;
						bestiBuff[storeloc+1] = i;
						bestiBuff[storeloc+2] = 8*rbidx + k;
						bestiBuff[storeloc+3] = threadID;
					}
				}
			}
		}
	}

}